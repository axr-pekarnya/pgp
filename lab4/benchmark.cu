#include "hip/hip_runtime.h"
#include <stdlib.h> 
#include <stdio.h> 
#include <stddef.h> 
#include <stdbool.h> 
#include <math.h> 
#include <iomanip>
#include <hip/hip_runtime.h>
#include <thrust/extrema.h> 
#include <thrust/device_vector.h> 

#define CSC(call) \
while (1) { \
    hipError_t status = call; \
    if (status != hipSuccess) { \
        printf("ERROR in %s:%d. Message: %s\n", __FILE__, __LINE__, hipGetErrorString(status)); \
        exit(0); \
    } \
    break; \
}

class TComparator { 
    public: 
        __host__ __device__ bool operator()(const double num1, const double num2) const { 
            return fabs(num1) < fabs(num2); 
        }
}; 

__global__ void SwapLines(double *matrix, double *unitedMatrix, int n, int i, int j) { 
    int posX = blockIdx.x * blockDim.x + threadIdx.x;  
    int shift = gridDim.x * blockDim.x; 
    double tmp;
    for (int k = posX; k < n; k += shift) { 
        tmp = matrix[n * k + i];
        matrix[n * k + i] = matrix[n * k + j];
        matrix[n * k + j] = tmp;
        tmp = unitedMatrix[n * k + i];
        unitedMatrix[n * k + i] = unitedMatrix[n * k + j];
        unitedMatrix[n * k + j] = tmp;
    }
} 

__global__ void Divide(double* matrix, double* unitedMatrix, int n) { 
    int posX = blockIdx.x * blockDim.x + threadIdx.x; 
    int posY = blockIdx.y * blockDim.y + threadIdx.y; 
    int shiftX = gridDim.x * blockDim.x; 
    int shiftY = gridDim.y * blockDim.y; 
    for (int i = posX; i < n; i += shiftX) { 
        for (int j = posY; j < n; j += shiftY) { 
            unitedMatrix[j * n + i] /= matrix[i * n + i]; 
        } 
    }  
} 

__global__ void DelLower(double* matrix, double* unitedMatrix, int n, int sep) { 
    int posX = blockIdx.x * blockDim.x + threadIdx.x; 
    int posY = blockIdx.y * blockDim.y + threadIdx.y; 
    int shiftX = gridDim.x * blockDim.x; 
    int shiftY = gridDim.y * blockDim.y; 
    for (int i = sep + 1 + posX; i < n; i += shiftX) { 
        double div = -matrix[sep * n + i] / matrix[sep * n + sep]; 
        for (int j = sep + 1 + posY; j < n; j += shiftY) { 
            matrix[j * n + i] += div * matrix[j * n + sep]; 
        } 
        for (int j = posY; j < n; j += shiftY) { 
            unitedMatrix[j * n + i] += div * unitedMatrix[j * n + sep]; 
        } 
    } 
} 

__global__ void DelUpper(double* matrix, double* unitedMatrix, int n, int sep) { 
    int posX = threadIdx.x + blockIdx.x * blockDim.x; 
    int posY = threadIdx.y + blockIdx.y * blockDim.y; 
    int shiftX = gridDim.x * blockDim.x; 
    int shiftY = gridDim.y * blockDim.y; 
    for (int i = sep - posX - 1; i >= 0; i -= shiftX) { 
        double div = -matrix[sep * n + i] / matrix[sep * n + sep]; 
        for (int j = posY; j < n; j += shiftY) { 
            unitedMatrix[j * n + i] += div * unitedMatrix[j * n + sep]; 
        } 
    } 
} 

void InvertMatrix(int n) {
    double* matrix = (double*)malloc(n * n * sizeof(double)); 
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            matrix[j * n + i] = rand() % 10 + 1; 
        }
    }
    double* unitedMatrix = (double*)malloc(n * n * sizeof(double)); 
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            unitedMatrix[i * n + j] = (i == j) ? 1 : 0;
        }
    }    
    double* devMatrix; 
    double* devUnitedMatrix; 
    hipMalloc(&devMatrix, sizeof(double) * n * n); 
    hipMalloc(&devUnitedMatrix, sizeof(double) * n * n); 
    hipMemcpy(devMatrix, matrix, sizeof(double) * n * n, hipMemcpyHostToDevice); 
    hipMemcpy(devUnitedMatrix, unitedMatrix, sizeof(double) * n * n, hipMemcpyHostToDevice); 
    dim3 block(32, 16); 
    dim3 thread(32, 16); 
    const thrust::device_ptr<double> ptr = thrust::device_pointer_cast(devMatrix); 
    const TComparator cmp; 
    hipEvent_t start, stop;
    float elapsedTime;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for (int i = 0; i < n - 1; ++i) { 
        const int maxIdx = thrust::max_element(ptr + i * n + i, ptr + (i + 1) * n, cmp) - ptr - i * n; 
        if (maxIdx != i){ 
            SwapLines<<<256, 256>>>(devMatrix, devUnitedMatrix, n, i, maxIdx); 
        } 
        DelLower<<<block, thread>>>(devMatrix, devUnitedMatrix, n, i); 
    }
    for (int i = n - 1; i > 0; i--) { 
        DelUpper<<<block, thread>>>(devMatrix, devUnitedMatrix, n, i); 
    } 
    Divide<<<block, thread>>>(devMatrix, devUnitedMatrix, n); 
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Size: %d x %d, Time: %f ms\n", n, n, elapsedTime);
    hipFree(devMatrix); 
    hipFree(devUnitedMatrix); 
    free(matrix); 
    free(unitedMatrix); 
}

int main() {
    InvertMatrix(3);
    //InvertMatrix(500);
    //InvertMatrix(1000);
    return 0;
}
